#include "hip/hip_runtime.h"
/*
 * This software is Copyright (c) 2011 Lukas Odzioba
 * <lukas dot odzioba at gmail dot com>
 * and it is hereby released to the general public under the following terms:
 * Redistribution and use in source and binary forms, with or without modification, are permitted.
 * Based on Alain Espinosa implementation http://openwall.info/wiki/john/MSCash
 */

#include <stdio.h>
#include "../cuda_mscash.h"
#include "cuda_common.cuh"

extern "C" void cuda_mscash(mscash_password *, mscash_hash *, mscash_salt *, int);

__constant__ mscash_salt cuda_salt[1];

__device__ static void md4_crypt(uint32_t * output, uint32_t * nt_buffer)
{
	unsigned int a = INIT_A;
	unsigned int b = INIT_B;
	unsigned int c = INIT_C;
	unsigned int d = INIT_D;

	/* Round 1 */
	a += (d ^ (b & (c ^ d))) + nt_buffer[0];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[1];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[2];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[3];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[4];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[5];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[6];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[7];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[8];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[9];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[10];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[11];
	b = (b << 19) | (b >> 13);

	a += (d ^ (b & (c ^ d))) + nt_buffer[12];
	a = (a << 3) | (a >> 29);
	d += (c ^ (a & (b ^ c))) + nt_buffer[13];
	d = (d << 7) | (d >> 25);
	c += (b ^ (d & (a ^ b))) + nt_buffer[14];
	c = (c << 11) | (c >> 21);
	b += (a ^ (c & (d ^ a))) + nt_buffer[15];
	b = (b << 19) | (b >> 13);

	/* Round 2 */
	a += ((b & (c | d)) | (c & d)) + nt_buffer[0] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[4] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[8] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[12] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[1] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[5] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[9] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[13] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[2] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[6] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[10] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[14] + SQRT_2;
	b = (b << 13) | (b >> 19);

	a += ((b & (c | d)) | (c & d)) + nt_buffer[3] + SQRT_2;
	a = (a << 3) | (a >> 29);
	d += ((a & (b | c)) | (b & c)) + nt_buffer[7] + SQRT_2;
	d = (d << 5) | (d >> 27);
	c += ((d & (a | b)) | (a & b)) + nt_buffer[11] + SQRT_2;
	c = (c << 9) | (c >> 23);
	b += ((c & (d | a)) | (d & a)) + nt_buffer[15] + SQRT_2;
	b = (b << 13) | (b >> 19);

	/* Round 3 */
	a += (d ^ c ^ b) + nt_buffer[0] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[8] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[4] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[12] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[2] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[10] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[6] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[14] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[1] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[9] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[5] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[13] + SQRT_3;
	b = (b << 15) | (b >> 17);

	a += (d ^ c ^ b) + nt_buffer[3] + SQRT_3;
	a = (a << 3) | (a >> 29);
	d += (c ^ b ^ a) + nt_buffer[11] + SQRT_3;
	d = (d << 9) | (d >> 23);
	c += (b ^ a ^ d) + nt_buffer[7] + SQRT_3;
	c = (c << 11) | (c >> 21);
	b += (a ^ d ^ c) + nt_buffer[15] + SQRT_3;
	b = (b << 15) | (b >> 17);

	output[0] = a + INIT_A;
	output[1] = b + INIT_B;
	output[2] = c + INIT_C;
	output[3] = d + INIT_D;
}

__device__ void prepare_key(uint16_t *key, int length, uint32_t *nt_buffer)
{
	int i = 0;
	for (i = 0; i < 16; i++)
		nt_buffer[i] = 0;
	for (i = 0; i < length / 2; i++)
		nt_buffer[i] = key[2 * i] | (key[2 * i + 1] << 16);
	if (length % 2 == 1)
		nt_buffer[i] = key[length - 1] | 0x800000;
	else
		nt_buffer[i] = 0x80;
	nt_buffer[14] = length << 4;
}

__device__ void prepare_login(uint16_t *login, int length,
    uint32_t *login_buffer)
{
	int i = 0;
	for (i = 0; i < 12; i++)
		login_buffer[i] = 0;
	for (i = 0; i < length / 2; i++)
		login_buffer[i] = login[2 * i] | (login[2 * i + 1] << 16);
	if (length % 2 == 1)
		login_buffer[i] = login[length - 1] | 0x800000;
	else
		login_buffer[i] = 0x80;
	login_buffer[10] = (length << 4) + 128;
}


__global__ void mscash_kernel(mscash_password * inbuffer,
    mscash_hash * outbuffer)
{
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint16_t *login = cuda_salt[0].salt;
	uint8_t loginlength = cuda_salt[0].length;

	uint16_t *password = inbuffer[idx].v;
	uint8_t passwordlength = inbuffer[idx].length;

	int i;
	uint32_t nt_buffer[16];
	uint32_t login_buffer[12];
	uint32_t output[4];

	prepare_key(password, passwordlength, nt_buffer);
	md4_crypt(output, nt_buffer);
	memcpy(nt_buffer, output, 4 * 4);

	prepare_login(login, loginlength, login_buffer);
	memcpy(nt_buffer + 4, login_buffer, 12 * 4);
	md4_crypt(output, nt_buffer);

	for (i = 0; i < 4; i++)
		outbuffer[idx].v[i] = output[i];

}

__host__ void cuda_mscash(mscash_password * inbuffer, mscash_hash * outbuffer,
                          mscash_salt *host_salt, int count)
{
	int blocks = (count + THREADS - 1) / THREADS;
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_salt), host_salt,
		sizeof(mscash_salt)));
	mscash_password *cuda_inbuffer;
	mscash_hash *cuda_outbuffer;

	size_t insize = sizeof(mscash_password) * KEYS_PER_CRYPT;
	size_t outsize = sizeof(mscash_hash) * KEYS_PER_CRYPT;

	HANDLE_ERROR(hipMalloc(&cuda_inbuffer, insize));
	HANDLE_ERROR(hipMalloc(&cuda_outbuffer, outsize));

	HANDLE_ERROR(hipMemcpy(cuda_inbuffer, inbuffer, insize,
		hipMemcpyHostToDevice));

	mscash_kernel <<< blocks, THREADS >>> (cuda_inbuffer, cuda_outbuffer);
	HANDLE_ERROR(hipGetLastError());

	HANDLE_ERROR(hipMemcpy(outbuffer, cuda_outbuffer, outsize,
		hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(cuda_inbuffer));
	HANDLE_ERROR(hipFree(cuda_outbuffer));

}
