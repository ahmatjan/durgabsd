#include "hip/hip_runtime.h"
/*
* This software is Copyright (c) 2011-2012 Lukas Odzioba <ukasz at openwall dot net>
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/
#ifndef _CUDA_COMMON_CU
#define _CUDA_COMMON_CU

#include <stdio.h>
#include <assert.h>

#include "../autoconfig.h"
#define HAVE_CUDA
#include "../common-gpu.h"
#include "hip/hip_runtime.h"
#include "cuda_common.cuh"

extern "C"
void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess) {
		fprintf(stderr, "%s in %s at line %d\n",
		    hipGetErrorString(err), file, line);
		if (err == hipErrorLaunchOutOfResources)
			fprintf(stderr, "Try decreasing THREADS in the corresponding hip/hip_runtime.h file. See doc/README-CUDA\n");
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

extern "C"
char *get_cuda_header_version()
{
	unsigned int minor=((CUDA_VERSION%100)/10)%10;
	unsigned int major=(CUDA_VERSION/1000)%100;
	static char ret[8];
	snprintf(ret,8,"%d.%d",major,minor);
	return ret;
}

extern "C"
static char *human_format(size_t size)
{
	char pref[] = { ' ', 'K', 'M', 'G', 'T', 'P', 'E', 'Z', 'Y' };
	int prefid = 0;
	static char ret[32];

	while (size > 1024) {
		size /= 1024;
		prefid++;
	}
	sprintf(ret, "%zd.%zd %cB", size, (size % 1024) / 100, pref[prefid]);
	return ret;
}

extern "C"
int cuda_id2nvml(int cuda_id)
{
#if __linux__ && HAVE_LIBDL
	hipDeviceProp_t devProp;
	hw_bus pci_info;

	hipGetDeviceProperties(&devProp, cuda_id);
	memset(pci_info.busId, 0, sizeof(pci_info.busId));
	sprintf(pci_info.busId, "%02x:%02x.%x",
	        devProp.pciBusID, devProp.pciDeviceID, devProp.pciDomainID);
	return id2nvml(pci_info);
#else
	return -1;
#endif
}

extern "C"
void cuda_device_list()
{
	int i, devices;
	hipError_t ret;
	int version;

	ret = hipGetDeviceCount(&devices);
	if (ret == hipErrorNoDevice) {
		puts("Error: No CUDA-capable devices were detected by the installed CUDA driver.\n");
		exit(EXIT_FAILURE);
	}
	if (ret == hipErrorInsufficientDriver) {
		puts("Error: The installed NVIDIA CUDA driver is older than the CUDA runtime library.\nThis is not a supported configuration. Update your display driver.\n");
		exit(EXIT_FAILURE);
	}
	if (hipRuntimeGetVersion(&version) == hipSuccess)
		printf("CUDA runtime %d.%d, ",
		       version / 1000, (version % 100) / 10);
	if (hipDriverGetVersion(&version) == hipSuccess)
		printf("driver %d.%d - ",
		       version / 1000, (version % 100) / 10);

	printf("%d CUDA device%s found:\n", devices, devices > 1 ? "s" : "");
	nvidia_probe();
	for (i = 0; i < devices; i++) {
		hipDeviceProp_t devProp;
		int arch_sm[] = { 1, 8, 32, 192, 0, 128 };

		hipGetDeviceProperties(&devProp, i);
		printf("\nCUDA Device #%d\n", i);
		printf("    Name:                          %s\n", devProp.name);
		printf("    Type:                          %s%s\n",
		    devProp.integrated ? "integrated" : "discrete",
		    devProp.tccDriver ? " (Tesla running tcc)" : "");
		printf("    Compute capability:            %d.%d (sm_%d%d)\n",
		       devProp.major, devProp.minor,
		       devProp.major, devProp.minor);

		if (devProp.major == 2 && devProp.minor >= 1)
		printf("    Number of stream processors:   %d (%d x %d)\n",
		    devProp.multiProcessorCount * 48,
		    devProp.multiProcessorCount, 48);
		if (devProp.major <= 5 && arch_sm[devProp.major])
		printf("    Number of stream processors:   %d (%d x %d)\n",
		    devProp.multiProcessorCount * arch_sm[devProp.major],
		    devProp.multiProcessorCount, arch_sm[devProp.major]);
		else /* We need to populate the arch_sm[] above */
		printf("    Number of multiprocessors:     %d\n",
		    devProp.multiProcessorCount);

		printf("    Clock rate:                    %d Mhz\n",
		    devProp.clockRate / 1000);
		printf("    Memory clock rate (peak)       %d Mhz\n",
		    devProp.memoryClockRate / 1000);
		printf("    Memory bus width               %d bits\n",
		    devProp.memoryBusWidth);
		printf("    Peak memory bandwidth:         %u GB/s\n",
		    2 * devProp.memoryClockRate *
		    (devProp.memoryBusWidth / 8) /
		    1000000);
		printf("    Total global memory:           %s%s\n",
		    human_format(devProp.totalGlobalMem),
		    devProp.ECCEnabled ? " (ECC)" : "");
		printf("    Total shared memory per block: %s\n",
		    human_format(devProp.sharedMemPerBlock));
		printf("    Total constant memory:         %s\n",
		    human_format(devProp.totalConstMem));

		if (devProp.l2CacheSize)
		printf("    L2 cache size                  %s\n",
		    human_format(devProp.l2CacheSize));
		else
		printf("    L2 cache:                      No\n");

		printf("    Kernel execution timeout:      %s\n",
		    (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
		printf("    Concurrent copy and execution: %s\n",
		    (devProp.asyncEngineCount == 2 ?
		     "Bi-directional" : devProp.asyncEngineCount == 1 ?
		     "One direction" : "No"));
		printf("    Concurrent kernels support:    %s\n",
		    (devProp.concurrentKernels ? "Yes" : "No"));
		printf("    Warp size:                     %d\n",
		    devProp.warpSize);
		printf("    Max. GPRs/thread block         %d\n",
		    devProp.regsPerBlock);
		printf("    Max. threads per block         %d\n",
		    devProp.maxThreadsPerBlock);
		printf("    Max. resident threads per MP   %d\n",
		    devProp.maxThreadsPerMultiProcessor);
		printf("    PCI device topology:           %02x:%02x.%x\n",
		    devProp.pciBusID, devProp.pciDeviceID, devProp.pciDomainID);
#if __linux__ && HAVE_LIBDL
		if (nvml_lib) {
			int fan, temp, util;
			int nvml_id = cuda_id2nvml(i);

			printf("    NVML id:                       %d\n",
			       nvml_id);
			fan = temp = util = -1;

			nvidia_get_temp(nvml_id, &temp, &fan, &util);
			if (fan >= 0)
				printf("    Fan speed:                     %d%%\n", fan);
			else
				printf("    Fan speed:                     n/a\n");
			if (temp >= 0)
				printf("    GPU temp:                      %d%sC\n",
				       temp, gpu_degree_sign);
			else
				printf("    GPU temp:                      n/a\n");
			if (util >= 0)
				printf("    Utilization:                   %d%%\n", util);
			else
				printf("    Utilization:                   n/a\n");
		}
#endif
		puts("");
	}
}

extern "C"
void *cuda_pageLockedMalloc(void *w, unsigned int size)
{
	HANDLE_ERROR(hipHostAlloc((void **) &w, size, hipHostMallocDefault));
	return w;
}

extern "C"
void cuda_pageLockedFree(void *w)
{
	HANDLE_ERROR(hipHostFree(w));
}

/* cuda init must be called first to set device */
extern "C"
int cuda_getAsyncEngineCount()
{
	hipDeviceProp_t prop;
	int dev;
	hipGetDevice(&dev);
	hipGetDeviceProperties(&prop,dev);
	return prop.asyncEngineCount;
	//if CUDA<4.0 we should use prop.overlapSupported
}
#endif
